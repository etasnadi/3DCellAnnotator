#include <stdio.h>
#include "hip/hip_runtime.h"

     



////////////////////////////////////////////////////////////////////////////////
// Cuda error checking
////////////////////////////////////////////////////////////////////////////////

void SAFE_CALL(hipError_t err){
	if(err != hipSuccess){
		printf("Error: %s \n", hipGetErrorString(err));
	}
}

void KERNEL_ERROR_CHECK(){
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if(errSync != hipSuccess){
		printf("\tSync kernel error: %s \n", hipGetErrorString(errSync));
	}
	if(errAsync != hipSuccess){
		printf("\tAsync kernel error: %s \n", hipGetErrorString(errAsync));
	}
}

void KERNEL_ERROR_CHECK(char const *message){
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if(errSync != hipSuccess){
		printf("%s\n", message);
		printf("\tSync kernel error: %s \n", hipGetErrorString(errSync));
	}
	if(errAsync != hipSuccess){
		printf("%s\n", message);
		printf("\tAsync kernel error: %s \n", hipGetErrorString(errAsync));
	}
}
