#include "cudaErr.cuh"

#include <iostream>




extern bool erro;

using namespace std;

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess)
	{
		std::cout << "Cuda error: " << hipGetErrorString(code) << " (" << file << ":" << line << ")" << std::endl;
		erro = true;
		//throw 20;
		//if (abort) exit(code);
	}
}
